#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "waveformat/waveformat.h"

#define TILE_SIZE 1024
#define MAX_MASK_WIDTH 5

typedef short element;

__global__ void convolution_1D_basic_kernel(element *N, element *P, element *M, int Mask_Width, int Width)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int N_start_point = i - (Mask_Width / 2);

    element Pvalue = 0;
    for (int j = 0; j < Mask_Width; j++)
    {
        if (N_start_point + j >= 0 && N_start_point + j < Width)
            Pvalue += N[N_start_point + j] * M[j];
    }
    P[i] = Pvalue / MAX_MASK_WIDTH; 
}

int main(int argc, char **argv)
{
    if (argc != 2)
    {
        printf("Please specify file name!\n");
        exit(EXIT_FAILURE);
    }
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float elapsedTime;

    element *data = NULL;
    element *data_clear = NULL;
    element *dev_data, *dev_data_clear;
    element *dev_M;

    const element h_M[MAX_MASK_WIDTH] = {1, 1, 1, 1, 1};
    hipMalloc((void**)&dev_M, sizeof(element) * MAX_MASK_WIDTH);

    hipMemcpy(dev_M, h_M, MAX_MASK_WIDTH * sizeof(element), hipMemcpyHostToDevice);

    int size;
    waveFormat fmt;
    FILE *f = NULL;
    
    f = fopen(argv[1], "rb");
    if (!f)
    {
        printf("Open input file failed!\n");
        return -1;
    }
    fmt = readWaveHeader(f);
    size = fmt.data_size;
    data = (element*)malloc(sizeof(element) * size);
    data_clear = (element*)malloc(sizeof(element) * size);

    fseek(f, 44L, SEEK_SET);
    fread(data, sizeof(element), size, f);
    if (f)
    {
        fclose(f);
        f = NULL;
    }
    
    hipMalloc((void**)&dev_data, sizeof(element) * size);
    hipMalloc((void**)&dev_data_clear, sizeof(element) * size);

    hipMemcpy(dev_data, data, sizeof(element) * size, hipMemcpyHostToDevice);

    dim3 block(TILE_SIZE, 1);
    dim3 grid((block.x - 1 + size) / block.x, 1);
    hipEventRecord(start, 0);
    convolution_1D_basic_kernel<<<grid, block>>>(dev_data, dev_data_clear, dev_M, MAX_MASK_WIDTH, size); 
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipMemcpy(data_clear, dev_data_clear, sizeof(element) * size, hipMemcpyDeviceToHost);

    f = fopen("audios/gpu_rst.wav", "wb+");
    if (!f)
    {
        printf("Open output file failed!\n");
        return -1;
    }
    writeWaveHeader(fmt, f);
    fseek(f, 44L, SEEK_SET);
    fwrite(data_clear, sizeof(element), size, f);

    fclose(f);
    f = NULL;
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("%s elapsed %f ms\n", argv[0], elapsedTime);
    
    free(data);
    free(data_clear);

    hipFree(dev_data);
    hipFree(dev_data_clear);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return 0;    
}
